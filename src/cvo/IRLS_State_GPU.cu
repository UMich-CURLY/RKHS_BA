#include "hip/hip_runtime.h"
#include "cvo/IRLS_State_GPU.hpp"
#include "cvo/IRLS_Cost_CPU.hpp"
#include "cvo/CvoFrameGPU.hpp"
#include "cvo/CvoFrame.hpp"
#include "cvo/CvoGPU_impl.cuh"
#include "cvo/CudaTypes.cuh"
#include "utils/CvoPointCloud.hpp"
#include "cvo/local_parameterization_se3.hpp"
#include "cupointcloud/cupointcloud.h"
#include "utils/data_type.hpp"
#include "cvo/SparseKernelMat.hpp"
#include <Eigen/Dense>
#include <Eigen/Sparse>


namespace cvo {

  BinaryStateGPU::BinaryStateGPU(std::shared_ptr<CvoFrameGPU> pc1,
                                 std::shared_ptr<CvoFrameGPU> pc2,
                                 const CvoParams * params_cpu,                                 
                                 const CvoParams * params_gpu,
                                 unsigned int num_neighbor,
                                 float init_ell
                                 ) : frame1_(pc1), frame2_(pc2),
                                     params_cpu_(params_cpu),
                                     params_gpu_(params_gpu),
                                     num_neighbors_(num_neighbor),
                                     ell_(static_cast<double>(init_ell)),
                                     init_num_neighbors_(num_neighbor){

    init_internal_SparseKernelMat_cpu(pc1->points->size(),  num_neighbor, &A_result_cpu_);
    A_device_ = init_SparseKernelMat_gpu(pc1->points->size(), num_neighbor, A_host_);
    clear_SparseKernelMat(&A_host_, num_neighbors_);

    is_optimizing_ell_ = params_cpu_->multiframe_is_optimizing_ell;
    if (is_optimizing_ell_) {
      num_neighbors_f1_ = num_neighbor;      
      A_f1_device_ = init_SparseKernelMat_gpu(pc1->points->size(), num_neighbor, A_f1_host_);
      init_internal_SparseKernelMat_cpu(pc1->points->size(),  num_neighbor, &A_f1_cpu_);
      clear_SparseKernelMat(&A_f1_host_, num_neighbors_f1_);
      
      num_neighbors_f2_ = num_neighbor;            
      A_f2_device_ = init_SparseKernelMat_gpu(pc2->points->size(), num_neighbor, A_f2_host_);
      init_internal_SparseKernelMat_cpu(pc2->points->size(),  num_neighbor, &A_f2_cpu_);      
      clear_SparseKernelMat(&A_f2_host_, num_neighbors_f2_);
    }
    //std::cout<<"Construct BinaryStateGPU: ell is "<<ell_<<", init_num_neighbors_ is "<<init_num_neighbors_<<"\n";

    if (params_cpu_->is_using_kdtree) {
      points_transformed_buffer_gpu_ = std::make_shared<CvoPointCloudGPU>(pc1->size());
      hipMalloc((int**)&cukdtree_inds_results_gpu_, sizeof(int)*init_num_neighbors_*pc1->size());
    }

    iter_ = 0;
    
    cudaMemGetInfoPrint(__func__);

  }

  BinaryStateGPU::~BinaryStateGPU() {
    delete_internal_SparseKernelMat_cpu(&A_result_cpu_);
    delete_SparseKernelMat_gpu(A_device_, &A_host_);
    if (params_cpu_->is_using_kdtree) {
      hipFree(cukdtree_inds_results_gpu_);
    }
    if (is_optimizing_ell_) {
      delete_SparseKernelMat_gpu(A_f1_device_, &A_f1_host_);
      delete_SparseKernelMat_gpu(A_f2_device_, &A_f2_host_);
    }
  }

  int BinaryStateGPU::update_inner_product() {

    unsigned int last_num_neibors = max_neighbors(&A_host_);
    if (last_num_neibors > 0)
      num_neighbors_ = std::min(init_num_neighbors_, (unsigned int)(last_num_neibors*1.1));
    clear_SparseKernelMat(&A_host_, num_neighbors_);    
    std::cout<< "Current num_neighbors_ is "<<num_neighbors_<<"\n";

    if (is_optimizing_ell_){
      unsigned int last_num_neibors_f1 = max_neighbors(&A_f1_host_);
      if (last_num_neibors_f1 > 0)
        num_neighbors_f1_ = std::min(init_num_neighbors_, (unsigned int)(last_num_neibors_f1*1.1));
      clear_SparseKernelMat(&A_f1_host_, num_neighbors_f1_);
    std::cout<< "Current num_neighbors_f1_ is "<<num_neighbors_f1_<<"\n";      
      
      unsigned int last_num_neibors_f2 = max_neighbors(&A_f2_host_);
      if (last_num_neibors_f2 > 0)
        num_neighbors_f2_ = std::min(init_num_neighbors_, (unsigned int)(last_num_neibors_f2*1.1));
      clear_SparseKernelMat(&A_f2_host_, num_neighbors_f2_);
    std::cout<< "Current num_neighbors_f2_ is "<<num_neighbors_f2_<<"\n";            
    }

    if (params_cpu_->is_using_kdtree) {

      thrust::device_vector<CvoPoint> & f1_points = frame1_->points_init_gpu()->points;
      thrust::device_vector<CvoPoint> & f2_points = frame2_->points_init_gpu()->points;
      
      //thrust::device_vector<int> cukdtree_inds_results;
      Eigen::Matrix4f T_f2_to_f1 = (frame2_->pose_cpu().inverse() * frame1_->pose_cpu()).cast<float>();
      // std::cout<<"T_f2_to_f1 is "<<T_f2_to_f1<<"\n";
      
      //thrust::device_ptr<int> inds_ptr_gpu = thrust::device_pointer_cast(cukdtree_inds_results_gpu_);
      //thrust::device_vector<int> inds_device_vec;
      //(inds_ptr_gpu, inds_ptr_gpu + num_neighbors_ * frame1_->size());
      hipMemset(cukdtree_inds_results_gpu_, -1, num_neighbors_ * frame1_->size()  * sizeof(int));

      //std::cout<<"find_nearby_source_points_cukdtree\n";
      find_nearby_source_points_cukdtree(//const CvoParams *cvo_params,
                                         frame1_->points_init_gpu(),
                                         frame2_->kdtree(),
                                         T_f2_to_f1,
                                         num_neighbors_,
                                         // output
                                         points_transformed_buffer_gpu_,
                                         cukdtree_inds_results_gpu_
                                         //inds_device_vec
                                         );

      /*
      thrust::device_ptr<int> inds_ptr_gpu_before = thrust::device_pointer_cast(cukdtree_inds_results_gpu_);
      thrust::device_vector<int> inds_device_vec_before(inds_ptr_gpu_before, inds_ptr_gpu_before + num_neighbors_ * frame1_->size());
      std::cout<<"First few indices before: ";
      for (int k = 0; k < num_neighbors_; k++) std::cout<<inds_device_vec_before[k]<<", ";
      std::cout<<"\n";
      */

      fill_in_A_mat_cukdtree<<< (f1_points.size() / CUDA_BLOCK_SIZE)+1, CUDA_BLOCK_SIZE  >>>
        (params_gpu_,
         thrust::raw_pointer_cast(points_transformed_buffer_gpu_->points.data()),         
         f1_points.size(),
         thrust::raw_pointer_cast(f2_points.data()),
         f2_points.size(),
         //thrust::raw_pointer_cast(inds_device_vec.data()),
         cukdtree_inds_results_gpu_,
         num_neighbors_, (float)ell_,
         A_device_);

      thrust::device_ptr<int> inds_ptr_gpu = thrust::device_pointer_cast(A_host_.ind_row2col);
      thrust::device_vector<int> inds_device_vec(inds_ptr_gpu, inds_ptr_gpu + num_neighbors_ * frame1_->size());
      thrust::device_ptr<float> A_ptr_gpu = thrust::device_pointer_cast(A_host_.mat);
      thrust::device_vector<float> A_device_vec(A_ptr_gpu, A_ptr_gpu + num_neighbors_ * frame1_->size());
      std::cout<<"First few indices after A mat: ";
      for (int k = 0; k < num_neighbors_; k++) std::cout<<"("<<inds_device_vec[k]<<": "<<A_device_vec[k]<<"), ";
      std::cout<<"\n";


    } else {

      fill_in_A_mat_gpu<<< (frame1_->points->size() / CUDA_BLOCK_SIZE)+1, CUDA_BLOCK_SIZE  >>>(
                                                                                               params_gpu_,
                                                                                               thrust::raw_pointer_cast(frame1_->points_transformed_gpu()->points.data()),
                                                                                               //frame1_->points_transformed_gpu(),
                                                                                               frame1_->points->size(),
                                                                                               thrust::raw_pointer_cast(frame2_->points_transformed_gpu()->points.data()),
                                                                                               //frame2_->points_transformed_gpu(),
                                                                                               frame2_->points->size(),
                                                                                               num_neighbors_,
                                                                                               (float)ell_,
                                                                                               A_device_
                                                                                               );
      if (is_optimizing_ell_) {
        fill_in_A_mat_gpu<<< (frame1_->points->size() / CUDA_BLOCK_SIZE)+1, CUDA_BLOCK_SIZE  >>>(
                                                                                                 params_gpu_,
                                                                                                 thrust::raw_pointer_cast(frame1_->points_transformed_gpu()->points.data()),
                                                                                                 //frame1_->points_transformed_gpu(),
                                                                                                 frame1_->points->size(),
                                                                                                 thrust::raw_pointer_cast(frame1_->points_transformed_gpu()->points.data()),
                                                                                                 //frame2_->points_transformed_gpu(),
                                                                                                 frame1_->points->size(),
                                                                                                 num_neighbors_f1_,
                                                                                                 (float)ell_,
                                                                                                 A_f1_device_
                                                                                                 );
        fill_in_A_mat_gpu<<< (frame2_->points->size() / CUDA_BLOCK_SIZE)+1, CUDA_BLOCK_SIZE  >>>(
                                                                                                 params_gpu_,
                                                                                                 thrust::raw_pointer_cast(frame2_->points_transformed_gpu()->points.data()),
                                                                                                 //frame1_->points_transformed_gpu(),
                                                                                                 frame2_->points->size(),
                                                                                                 thrust::raw_pointer_cast(frame2_->points_transformed_gpu()->points.data()),
                                                                                                 //frame2_->points_transformed_gpu(),
                                                                                                 frame2_->points->size(),
                                                                                                 num_neighbors_f2_,
                                                                                                 (float)ell_,
                                                                                                 A_f2_device_);
      
      }
      /*
      thrust::device_ptr<int> inds_ptr_gpu = thrust::device_pointer_cast(A_host_.ind_row2col);
      thrust::device_vector<int> inds_device_vec(inds_ptr_gpu, inds_ptr_gpu + num_neighbors_ * frame1_->size());
      std::cout<<"First few indices: ";
      for (int k = 0; k < num_neighbors_; k++) std::cout<<inds_device_vec[k]<<", ";
      std::cout<<"\n";
      */
      hipDeviceSynchronize();    
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) { 
        fprintf(stderr, "IRLS_State_GPU.cu: fill_in_A_mat_gpu: Failed to run fill_in_A_mat_gpu %s .\n", hipGetErrorString(err)); 
        exit(EXIT_FAILURE); 
      }
      
    }
    compute_nonzeros(&A_host_);
    std::cout<<"Nonzeros is "<<A_host_.nonzero_sum<<"\n";
    if (is_optimizing_ell_) {
      compute_nonzeros(&A_f1_host_);
      std::cout<<"Nonzeros f1 is "<<A_f1_host_.nonzero_sum<<"\n";
      compute_nonzeros(&A_f2_host_);
      std::cout<<"Nonzeros f2 is "<<A_f2_host_.nonzero_sum<<"\n";
      
    }

    
    if (A_host_.nonzero_sum == 0) {
      cvo::Mat44d_row T1_ = cvo::Mat44d_row::Identity();
      cvo::Mat44d_row T2_ = cvo::Mat44d_row::Identity();
      T1_.block(0,0,3,4) = Eigen::Map<Eigen::Matrix<double, 3, 4, Eigen::RowMajor>>(frame1_->pose_vec);
      T2_.block(0,0,3,4) = Eigen::Map<Eigen::Matrix<double, 3, 4, Eigen::RowMajor>>(frame2_->pose_vec);

      //Sophus::SE3d T1_s(T1_.block(0,0,3,3), T1_.block(0,3,3,1));
      //Sophus::SE3d T2_s(T2_.block(0,0,3,3), T2_.block(0,3,3,1));
      //if ((T1_s.inverse() * T2_s).log().norm() < 3  ) {
        
      //}
      
      // Eigen::Map<Eigen::Matrix<double, 3, 4, Eigen::RowMajor>>
      std::cout<<"Edge state ptr "<<this<<" is zero\n";
	std::cout<<"T1:\n "<<Eigen::Map<Eigen::Matrix<double, 3, 4, Eigen::RowMajor>>(frame1_->pose_vec)<<"\n";
	std::cout<<"T2:\n "<<Eigen::Map<Eigen::Matrix<double, 3, 4, Eigen::RowMajor>>(frame2_->pose_vec)<<"\n";
	//std::cout<<"f1 #: "<<frame1_->points_transformed_gpu()->points[0].getVector3fMap()<<"\n";
	//std::cout<<"f2 #: "<<frame2_->points_transformed_gpu()->points[0].getVector3fMap()<<"\n";
    }

    copy_internal_SparseKernelMat_gpu_to_cpu(&A_host_, &A_result_cpu_,
                                             num_neighbors_);
    if (params_cpu_->multiframe_is_optimizing_ell) {
      copy_internal_SparseKernelMat_gpu_to_cpu(&A_f1_host_, &A_f1_cpu_,
                                               num_neighbors_f1_);
      copy_internal_SparseKernelMat_gpu_to_cpu(&A_f2_host_, &A_f2_cpu_,
                                               num_neighbors_f2_);
      
    }
    
    cudaMemGetInfoPrint(__func__);    

    iter_++;    
    return A_result_cpu_.nonzero_sum;;
    //if (ip_mat_.nonZeros() < 100) {
    //  std::cout<<"too sparse inner product mat "<<ip_mat_.nonZeros()<<std::endl;
    //  return -1;
    //} else
    //  return 0;
    
  }

  const CvoFrame * BinaryStateGPU::get_frame1() const {
    return frame1_.get();
  }
  const CvoFrame * BinaryStateGPU::get_frame2() const {
    return frame2_.get();
  }

  void BinaryStateGPU::export_association(Association & output) {
    gpu_association_to_cpu(A_host_,
                           output,
                           frame1_->points->size(),
                           frame2_->points->size(),
                           num_neighbors_);
  }

  
}

